#include "hip/hip_runtime.h"
// cuda_kernels.cu
#include "cuda_kernels.cuh"
#include <iostream>
#include <stdio.h>


// Kernel for normal memory mode
__global__ void matMulKernelNormal(bool* A, bool* B, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
    
    if (row < height && col < width) {
        int count = 0;
        // Count alive neighbors
        for(int i = -1; i <= 1; i++) {
            for(int j = -1; j <= 1; j++) {
                if(i == 0 && j == 0) continue;
                int newRow = row + i;
                int newCol = col + j;
                if(newRow >= 0 && newRow < height && newCol >= 0 && newCol < width) {
                    if(A[newRow * width + newCol]) count++;
                }
            }
        }
        
        // Apply rules
        if(count == 3) {
            B[index] = true;
        } else if(count == 2 && A[index]) {
            B[index] = true;
        } else {
            B[index] = false;
        }
    }
}

__global__ void matMulKernel(Matrix* A, Matrix* B, int width, int height) {
    // get position of current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
   
    // update matrix
    if (row < height && col < width) {
        int aliveNeighbors = countAliveMembers(A, row, col);
        // check rules and generate matrix after update
        if( aliveNeighbors == 3){
            B->elements[index] = true;
        }else if( aliveNeighbors == 2 && A->elements[index]){
            B->elements[index] = true;
        }else{
            B->elements[index] = false;
        }
    }
    // now matrix B is the next frame
}

__global__ void testKernel(bool *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = !data[idx];  // flip the value
    }
}

void launchMatMulKernel(Matrix* A, Matrix* B, bool* d_A, bool* d_B, int width, int height, std::string processingType) {
    // int blockDim = (int)sqrt(numThreads);
    int blockDim = 32;
    std::cout << "blockDim: " << blockDim << std::endl;
    dim3 blockSize(blockDim, blockDim);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);

    // testKernel<<<gridSize, blockSize>>>(A->elements, width * height);
    
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        std::cout << "Test kernel error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();

    if( processingType == "NORMAL" ){
        // copy data to device  

        // A is the current frame
        hipMemcpy(d_A, A->elements, width * height * sizeof(bool), hipMemcpyHostToDevice);

        matMulKernelNormal<<<gridSize, blockSize>>>(d_A, d_B, width, height);

        hipDeviceSynchronize();

        // copy data to host
        hipMemcpy(B->elements, d_B, width * height * sizeof(bool), hipMemcpyDeviceToHost);
        
    }else{
        // check the number of white cells in A
        int countA = 0;
        for(int i = 0; i < width * height; i++) {
            if(A->elements[i]) countA++;
        }
        std::cout << "Before kernel - white cells in A: " << countA << std::endl;
        
        // check the matrix dimensions
        std::cout << "Matrix dimensions - Width: " << A->width << ", Height: " << A->height << std::endl;
        std::cout << "Grid dimensions - x: " << gridSize.x << ", y: " << gridSize.y << std::endl;
        std::cout << "Block dimensions - x: " << blockSize.x << ", y: " << blockSize.y << std::endl;

        // check the CUDA error
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cout << "CUDA error before kernel: " << hipGetErrorString(err) << std::endl;
        }

        matMulKernel<<<gridSize, blockSize>>>(A, B, width, height);

        // check the kernel execution error
        err = hipGetLastError();
        if(err != hipSuccess) {
            std::cout << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
        }

        // check the number of white cells in B
        hipDeviceSynchronize();
        int count = 0;
        for( int i = 0; i < width * height; i++){
            if( B->elements[i]){
                count++;
            }
        }
        std::cout << "current white count: " << count << std::endl;
    }
}

__device__ int countAliveMembers(Matrix *A, int row, int col) {
    int count = 0;
    // iterate all neighbors
    for( int i = -1; i <= 1; i++){
        for( int j = -1; j <= 1; j++){
            // skip itself    
            if( i == 0 && j == 0){
                continue;
            }
            // count the number of alive neighbors
            if(row + i >= 0 && row + i < A->height && col + j >= 0 && col + j < A->width){
                if(A->elements[(row + i) * A->width + col + j]){
                    count++;
                }
            }
        }
    }
    return count;
}
