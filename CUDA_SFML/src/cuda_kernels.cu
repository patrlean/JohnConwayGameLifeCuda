#include "hip/hip_runtime.h"
// cuda_kernels.cu
#include "cuda_kernels.cuh"
#include <iostream>
#include <stdio.h>


// Kernel for normal memory mode
__global__ void matMulKernelNormal(bool* A, bool* B, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
    
    if (row < height && col < width) {
        int count = 0;
        // Count alive neighbors
        for(int i = -1; i <= 1; i++) {
            for(int j = -1; j <= 1; j++) {
                if(i == 0 && j == 0) continue;
                int newRow = row + i;
                int newCol = col + j;
                if(newRow >= 0 && newRow < height && newCol >= 0 && newCol < width) {
                    if(A[newRow * width + newCol]) count++;
                }
            }
        }
        
        // Apply rules
        if(count == 3) {
            B[index] = true;
        } else if(count == 2 && A[index]) {
            B[index] = true;
        } else {
            B[index] = false;
        }
    }
}

__global__ void matMulKernel(Matrix* A, Matrix* B, int width, int height) {
    // get position of current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
   
    // update matrix
    if (row < height && col < width) {
        int aliveNeighbors = countAliveMembers(A, row, col);
        // check rules and generate matrix after update
        if( aliveNeighbors == 3){
            setElement(B, row, col, true);
        }else if( aliveNeighbors == 2 && A->elements[index]){
            setElement(B, row, col, true);
        }else{
            setElement(B, row, col, false);
        }
    }
    // now matrix B is the next frame
}

void launchMatMulKernel(Matrix* A, Matrix* B, bool* d_A, bool* d_B, int width, int height, std::string processingType) {
    int blockDim = (int)sqrt(numThreads);
    dim3 blockSize(blockDim, blockDim);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);

    if( processingType == "NORMAL" ){
        // copy data to device  

        // A is the current frame
        hipMemcpy(d_A, A->elements, width * height * sizeof(bool), hipMemcpyHostToDevice);

        matMulKernelNormal<<<gridSize, blockSize>>>(d_A, d_B, width, height);

        // copy data to host
        hipMemcpy(B->elements, d_B, width * height * sizeof(bool), hipMemcpyDeviceToHost);

        // free device memory
        hipFree(d_A);
        hipFree(d_B);
    }else{
        matMulKernel<<<gridSize, blockSize>>>(A, B, width, height);
    }
}


__device__ void setElement(Matrix *A, int row, int col, bool value) {
    A->elements[row * A->width + col] = value;
}

__device__ int countAliveMembers(Matrix *A, int row, int col) {
    int count = 0;
    // iterate all neighbors
    for( int i = -1; i <= 1; i++){
        for( int j = -1; j <= 1; j++){
            // skip itself    
            if( i == 0 && j == 0){
                continue;
            }
            // count the number of alive neighbors
            if(row + i >= 0 && row + i < A->height && col + j >= 0 && col + j < A->width){
                if(A->elements[(row + i) * A->width + col + j]){
                    count++;
                }
            }
        }
    }
    return count;
}
