#include "hip/hip_runtime.h"
// cuda_kernels.cu
#include "cuda_kernels.cuh"
#include <iostream>
#include <stdio.h>

__global__ void matMulKernel(Matrix* A, Matrix* B, int width, int height) {
    // get position of current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
    // printf(" thread x index is %d\n", threadIdx.x);
    // print element
    if (row == 400 && col == 200){
       printf("element is %d\n", A -> elements[index]);
       }
    
    // update matrix
    if (row < height && col < width) {
        int aliveNeighbors = countAliveMembers(A, row, col);
        // check rules and generate update matrix
        if( aliveNeighbors == 3 && !A->elements[index]){
            setElement(B, row, col, true);
        }else if( (aliveNeighbors != 2 && aliveNeighbors != 3) && A->elements[index]){
            setElement(B, row, col, false);
        }else{
            setElement(B, row, col, false);
        }
    }
}

void launchMatMulKernel(Matrix* A, Matrix* B, int width, int height) {
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);
    matMulKernel<<<gridSize, blockSize>>>(A, B, width, height)
}


__device__ void setElement(Matrix *A, int row, int col, bool value) {
    A->elements[row * A->width + col] = value;
}

__device__ int countAliveMembers(Matrix *A, int row, int col) {
    int count = 0;
    // iterate all neighbors
    for( int i = -1; i <= 1; i++){
        for( int j = -1; j <= 1; j++){
            // skip itself    
            if( i == 0 && j == 0){
                continue;
            }
            // count the number of alive neighbors
            if(row + i >= 0 && row + i < A->height && col + j >= 0 && col + j < A->width){
                if(A->elements[(row + i) * A->width + col + j]){
                    count++;
                }
            }
        }
    }
    return count;
}
