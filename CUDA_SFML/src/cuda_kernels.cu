#include "hip/hip_runtime.h"
/*
Author: Tianyou Zhao 
Class: ECE6122 
Last Date Modified: 03-11-2024
Description:
This is the implementation of cuda_kernels.cuh, which is used to implement the CUDA kernels.
*/
#include "cuda_kernels.cuh"
#include <iostream>
#include <stdio.h>
static hipStream_t stream1, stream2; 
static bool streamsCreated = false;
static bool* device_A1 = nullptr;
static bool* device_B1 = nullptr;
static bool* device_A2 = nullptr;
static bool* device_B2 = nullptr;
static hipEvent_t event1, event2;
// Kernel for normal memory mode
// input: A is the current frame, B is the next frame
// output: B is the next frame
// Normal mode for memory management
__global__ void matMulKernelNormal(bool* A, bool* B, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
    
    if (row < height && col < width) {
        int count = 0;
        // Count alive neighbors
        for(int i = -1; i <= 1; i++) {
            for(int j = -1; j <= 1; j++) {
                if(i == 0 && j == 0) continue;
                int newRow = row + i;
                int newCol = col + j;
                if(newRow >= 0 && newRow < height && newCol >= 0 && newCol < width) {
                    if(A[newRow * width + newCol]) count++;
                }
            }
        }
        
        // Apply rules
        if(count == 3) {
            B[index] = true;
        } else if(count == 2 && A[index]) {
            B[index] = true;
        } else {
            B[index] = false;
        }
    }
}

// Kernel for pinned memory mode and managed memory mode
// input: A is the current frame, B is the next frame
// output: B is the next frame
__global__ void matMulKernel(Matrix* A, Matrix* B, int width, int height) {
    // get position of current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
   
    // update matrix
    if (row < height && col < width) {
        int aliveNeighbors = countAliveMembers(A, row, col);
        // check rules and generate matrix after update
        if( aliveNeighbors == 3){
            B->elements[index] = true;
        }else if( aliveNeighbors == 2 && A->elements[index]){
            B->elements[index] = true;
        }else{
            B->elements[index] = false;
        }
    }
    // now matrix B is the next frame
}

// Kernel for testing
__global__ void testKernel(bool *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = !data[idx];  // flip the value
    }
}

// launch the kernel
// input: A is the current frame, B is the next frame, d_A and d_B are the device memory for A and B, 
//        width and height are the dimensions of the matrix, processingType is the type of memory management
// output: B is the next frame
void launchMatMulKernel(Matrix* A, Matrix* B, bool* d_A, bool* d_B, int width, int height, std::string processingType) {
    // define the block size and grid size
    int blockDim = (int)sqrt(numThreads) + 1;
    dim3 blockSize(blockDim, blockDim);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);
    
    // launch the kernel
    if( processingType == "NORMAL" ){
        // copy data to device  
        hipMemcpy(d_A, A->elements, width * height * sizeof(bool), hipMemcpyHostToDevice);
        // launch the kernel
        matMulKernelNormal<<<gridSize, blockSize>>>(d_A, d_B, width, height);
        hipDeviceSynchronize();
        // copy data to host
        hipMemcpy(B->elements, d_B, width * height * sizeof(bool), hipMemcpyDeviceToHost);
    }else if( processingType == "PINNED"){
        // calculate the actual size of each half
        int halfHeight = height / 2;
        int remainingHeight = height - halfHeight; // handle odd height
        
        // calculate the actual size of each part (in bytes)
        int size1 = halfHeight * width * sizeof(bool);
        int size2 = remainingHeight * width * sizeof(bool);

        if (!streamsCreated) {
            // create CUDA stream and event
            hipStreamCreate(&stream1);
            hipStreamCreate(&stream2);
            hipEventCreate(&event1);
            hipEventCreate(&event2);
            
            // allocate device memory for two streams
            hipMalloc((void**)&device_A1, size1);
            hipMalloc((void**)&device_B1, size1);
            hipMalloc((void**)&device_A2, size2);
            hipMalloc((void**)&device_B2, size2);
            
            // copy data to device
            hipMemcpyAsync(device_A1, A->elements, size1, 
                          hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(device_A2, A->elements + (halfHeight * width), 
                          size2, hipMemcpyHostToDevice, stream2);
            
            // wait for initial data transfer to complete
            hipStreamSynchronize(stream1);
            hipStreamSynchronize(stream2);
            
            streamsCreated = true;
        }
        // calculate the grid size
        dim3 gridSize1((width + blockSize.x - 1) / blockSize.x, 
                      (halfHeight + blockSize.y - 1) / blockSize.y);
        dim3 gridSize2((width + blockSize.x - 1) / blockSize.x, 
                      (remainingHeight + blockSize.y - 1) / blockSize.y);
        // launch the kernel in two streams
        matMulKernelNormal<<<gridSize1, blockSize, 0, stream1>>>(
            device_A1, device_B1, width, halfHeight);
        matMulKernelNormal<<<gridSize2, blockSize, 0, stream2>>>(
            device_A2, device_B2, width, remainingHeight);
        // record event
        hipEventRecord(event1, stream1);
        hipEventRecord(event2, stream2);
        // wait for kernel to complete
        hipEventSynchronize(event1);
        hipEventSynchronize(event2);
        // swap pointers
        bool* temp1 = device_A1;
        device_A1 = device_B1;
        device_B1 = temp1;
        bool* temp2 = device_A2;
        device_A2 = device_B2;
        device_B2 = temp2;
        // copy data to host
        hipMemcpyAsync(B->elements, device_B1, size1, 
                       hipMemcpyDeviceToHost, stream1);
        hipMemcpyAsync(B->elements + (halfHeight * width), device_B2, 
                       size2, hipMemcpyDeviceToHost, stream2);
        // use event to synchronize
        hipEventSynchronize(event1);
        hipEventSynchronize(event2);
    }else if( processingType == "MANAGED"){
        matMulKernel<<<gridSize, blockSize>>>(A, B, width, height);
        hipDeviceSynchronize();
    }
}

// count the number of alive neighbors
// input: A is the current frame, row and col are the position of the cell
// output: the number of alive neighbors
__device__ int countAliveMembers(Matrix *A, int row, int col) {
    int count = 0;
    // iterate all neighbors
    for( int i = -1; i <= 1; i++){
        for( int j = -1; j <= 1; j++){
            // skip itself    
            if( i == 0 && j == 0){
                continue;
            }
            // count the number of alive neighbors
            if(row + i >= 0 && row + i < A->height && col + j >= 0 && col + j < A->width){
                if(A->elements[(row + i) * A->width + col + j]){
                    count++;
                }
            }
        }
    }
    return count;
}
