#include "hip/hip_runtime.h"
/*
Author: Tianyou Zhao 
Class: ECE6122 
Last Date Modified: 03-11-2024
Description:
This is the implementation of cuda_kernels.cuh, which is used to implement the CUDA kernels.
*/
#include "cuda_kernels.cuh"
#include <iostream>
#include <stdio.h>


// Kernel for normal memory mode
// input: A is the current frame, B is the next frame
// output: B is the next frame
// Normal mode for memory management
__global__ void matMulKernelNormal(bool* A, bool* B, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
    
    if (row < height && col < width) {
        int count = 0;
        // Count alive neighbors
        for(int i = -1; i <= 1; i++) {
            for(int j = -1; j <= 1; j++) {
                if(i == 0 && j == 0) continue;
                int newRow = row + i;
                int newCol = col + j;
                if(newRow >= 0 && newRow < height && newCol >= 0 && newCol < width) {
                    if(A[newRow * width + newCol]) count++;
                }
            }
        }
        
        // Apply rules
        if(count == 3) {
            B[index] = true;
        } else if(count == 2 && A[index]) {
            B[index] = true;
        } else {
            B[index] = false;
        }
    }
}

// Kernel for pinned memory mode and managed memory mode
// input: A is the current frame, B is the next frame
// output: B is the next frame
__global__ void matMulKernel(Matrix* A, Matrix* B, int width, int height) {
    // get position of current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
   
    // update matrix
    if (row < height && col < width) {
        int aliveNeighbors = countAliveMembers(A, row, col);
        // check rules and generate matrix after update
        if( aliveNeighbors == 3){
            B->elements[index] = true;
        }else if( aliveNeighbors == 2 && A->elements[index]){
            B->elements[index] = true;
        }else{
            B->elements[index] = false;
        }
    }
    // now matrix B is the next frame
}

// Kernel for testing
__global__ void testKernel(bool *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = !data[idx];  // flip the value
    }
}

// launch the kernel
// input: A is the current frame, B is the next frame, d_A and d_B are the device memory for A and B, 
//        width and height are the dimensions of the matrix, processingType is the type of memory management
// output: B is the next frame
void launchMatMulKernel(Matrix* A, Matrix* B, bool* d_A, bool* d_B, int width, int height, std::string processingType) {
    // define the block size and grid size
    int blockDim = (int)sqrt(numThreads) + 1;
    dim3 blockSize(blockDim, blockDim);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);

    // launch the kernel
    if( processingType == "NORMAL" ){
        // copy data to device  
        hipMemcpy(d_A, A->elements, width * height * sizeof(bool), hipMemcpyHostToDevice);
        // launch the kernel
        matMulKernelNormal<<<gridSize, blockSize>>>(d_A, d_B, width, height);
        hipDeviceSynchronize();
        // copy data to host
        hipMemcpy(B->elements, d_B, width * height * sizeof(bool), hipMemcpyDeviceToHost);
    }else if( processingType == "PINNED"){
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        // copy data to device
        hipMemcpyAsync(d_A, A->elements, width * height * sizeof(bool), hipMemcpyHostToDevice, stream1);
        // run kernel in stream1
        matMulKernelNormal<<<gridSize, blockSize, 0, stream1>>>(d_A, d_B, width, height);
        // copy data to host
        hipMemcpyAsync(B->elements, d_B, width * height * sizeof(bool), hipMemcpyDeviceToHost, stream1);
        // synchronize stream
        hipStreamSynchronize(stream1);

    }else if( processingType == "MANAGED"){
        matMulKernel<<<gridSize, blockSize>>>(A, B, width, height);
        hipDeviceSynchronize();
    }
}

// count the number of alive neighbors
// input: A is the current frame, row and col are the position of the cell
// output: the number of alive neighbors
__device__ int countAliveMembers(Matrix *A, int row, int col) {
    int count = 0;
    // iterate all neighbors
    for( int i = -1; i <= 1; i++){
        for( int j = -1; j <= 1; j++){
            // skip itself    
            if( i == 0 && j == 0){
                continue;
            }
            // count the number of alive neighbors
            if(row + i >= 0 && row + i < A->height && col + j >= 0 && col + j < A->width){
                if(A->elements[(row + i) * A->width + col + j]){
                    count++;
                }
            }
        }
    }
    return count;
}
